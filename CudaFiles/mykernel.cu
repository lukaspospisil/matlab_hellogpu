
#include <hip/hip_runtime.h>


__global__ void mykernel( double *aplusb, 
						  const double *a, 
						  const double *b, 
						  const int N  
                ) {
	/* "const" because the value is not changing in kernel, N = size of vectors */

	// compute index of this kernel
	int n = blockIdx.x*blockDim.x + threadIdx.x;

	// if index is smaller than size, then compute something
	if(n<N){
		aplusb[n] = a[n] + b[n];
	} else {
		/* put your feet on the table */
	}

}
